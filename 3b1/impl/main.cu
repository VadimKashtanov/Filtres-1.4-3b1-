#include "main.cuh"

#include "../impl_tmpl/tmpl_etc.cu"

static void plume_pred(Mdl_t * mdl, uint t0, uint t1) {
	float * ancien = mdl_pred(mdl, t0, t1, 3);
	printf("PRED GENERALE = ");
	FOR(0, p, P) printf(" %f%% ", 100*ancien[p]);
	printf("\n");
	free(ancien);
};

float pourcent_masque_nulle[C] = {0};

float * pourcent_masque = de_a(0.80, 0.0, C);

//	! A FAIRE ! :
//		selection (mutation de +/- 1 ligne (de meme source))
//

float * alpha = de_a(1e-3, 1e-3, C);

PAS_OPTIMISER()
int main(int argc, char ** argv) {
	MSG("S(x) Ajouter un peut d'aléatoire");
	MSG("S(x) Eventuellement faire des prediction plus lointaines");
	//	-- Init --
	srand(0);
	hipSetDevice(0);
	titre(" Charger tout ");   charger_tout();

	//	-- Verification --
	//titre("Verifier MDL");     verif_mdl_1e5();

	//===============
	titre("  Programme Generale  ");

	uint Y[C] = {
		1024,
		512,512,
		256,256,
		128,128,
		64,
		32,
		16,
		8,
		4,
		P
	};
	uint insts[C] = {
		FILTRES_PRIXS,
		DOT1D,DOT1D,
		DOT1D,DOT1D,
		DOT1D,DOT1D,
		DOT1D,
		DOT1D,
		DOT1D,
		DOT1D,
		DOT1D,
		DOT1D
	};
	//
	uint lignes[BLOQUES] = {0};
	FOR(0, i, BLOQUES) lignes[i] = rand() % EMA_INTS;
	//
	uint decales[BLOQUES] = {0};
	FOR(0, i, BLOQUES) decales[i] = rand() % MAX_DECALES;
	//	Assurances :
	FOR(0, i, 3) {
		lignes [i] = 0;
		decales[i] = 0;
	}
	//
	Mdl_t * mdl = cree_mdl(Y, insts, lignes, decales);

	//Mdl_t * mdl = ouvrire_mdl("mdl.bin");

	plumer_mdl(mdl);

	//	================= Initialisation ==============
	uint t0 = DEPART;
	uint t1 = ROND_MODULO(FIN, (16*16));
	printf("t0=%i t1=%i FIN=%i (t1-t0=%i, %%(16*16)=%i)\n", t0, t1, FIN, t1-t0, (t1-t0)%(16*16));
	//
	plume_pred(mdl, t0, t1);
	//
	uint REP = 150;
	FOR(0, rep, REP) {
		FOR(0, i, 5) {
			printf(" ================== %i/20 ================\n", i);
			optimisation_mini_packet(
				mdl,
				t0, t1, 16*16*1,
				alpha, 1.0,
				RMSPROP, 70,
				pourcent_masque);
			plume_pred(mdl, t0, t1);
			mdl_gpu_vers_cpu(mdl);
			ecrire_mdl(mdl, "mdl.bin");
		}
		//
		optimiser(
			mdl,
			t0, t1,
			alpha, 1.0,
			RMSPROP, 1000,
			pourcent_masque_nulle);
		//
		mdl_gpu_vers_cpu(mdl);
		ecrire_mdl(mdl, "mdl.bin");
		plume_pred(mdl, t0, t1);
		printf("===================================================\n");
		printf("==================TERMINE %i/%i=======================\n", rep+1, REP);
		printf("===================================================\n");
	}
	//
	mdl_gpu_vers_cpu(mdl);
	ecrire_mdl(mdl, "mdl.bin");
	liberer_mdl(mdl);

	//	-- Fin --
	liberer_tout();
};