#include "mdl.cuh"

//	Sources
float   prixs[PRIXS] = {};
float   macds[PRIXS] = {};
float volumes[PRIXS] = {};
float   hight[PRIXS] = {};
float    low[PRIXS] = {};
//
float            ema[EMA_INTS * PRIXS *    1  ] = {};
float     normalisee[EMA_INTS * PRIXS * N_FLTR] = {};
float dif_normalisee[EMA_INTS * PRIXS * N_FLTR] = {};

void charger_les_prixs() {
	uint __PRIXS;
	FILE * fp;
	//
	fp = fopen("prixs/prixs.bin", "rb");
	ASSERT(fp != 0);
	(void)!fread(&__PRIXS, sizeof(uint), 1, fp);
	ASSERT(__PRIXS == PRIXS);
	(void)!fread(prixs, sizeof(float), PRIXS, fp);
	fclose(fp);
	//
	fp = fopen("prixs/volumes.bin", "rb");
	ASSERT(fp != 0);
	(void)!fread(&__PRIXS, sizeof(uint), 1, fp);
	ASSERT(__PRIXS == PRIXS);
	(void)!fread(volumes, sizeof(float), PRIXS, fp);
	fclose(fp);
	//
	fp = fopen("prixs/macds.bin", "rb");
	ASSERT(fp != 0);
	(void)!fread(&__PRIXS, sizeof(uint), 1, fp);
	ASSERT(__PRIXS == PRIXS);
	(void)!fread(macds, sizeof(float), PRIXS, fp);
	fclose(fp);
	//
	fp = fopen("prixs/hight.bin", "rb");
	ASSERT(fp != 0);
	(void)!fread(&__PRIXS, sizeof(uint), 1, fp);
	ASSERT(__PRIXS == PRIXS);
	(void)!fread(hight, sizeof(float), PRIXS, fp);
	fclose(fp);
	//
	fp = fopen("prixs/low.bin", "rb");
	ASSERT(fp != 0);
	(void)!fread(&__PRIXS, sizeof(uint), 1, fp);
	ASSERT(__PRIXS == PRIXS);
	(void)!fread(low, sizeof(float), PRIXS, fp);
	fclose(fp);
};

void calculer_ema_norm_diff() {
	//	extern float        ema[EMA_INTS][    PRIXS    ];
	float k[EMA_INTS];
	float _k[EMA_INTS];
	for (uint i=0; i < EMA_INTS; i++) {
		k[i] = 1.0/(1.0 + (float)ema_ints[i].ema);
		_k[i] = 1.0 - k[i];
		ema[i*PRIXS+0] = (ema_ints[i].source)[0];

		assert(ema_ints[i].interv <= MAX_INTERVALLE);
	}
	//
	for (uint i=1; i < PRIXS; i++) {
		for (uint j=0; j < EMA_INTS; j++) {
			ema[j*PRIXS+i] = (ema_ints[j].source)[i]*k[j] + ema[j*PRIXS + i-1]*_k[j];
		};
	};


	//	extern float normalisee[EMA_INTS][PRIXS][N_FLTR];
	float _max, _min;
	FOR(DEPART, t, FIN) {
		FOR(0, e, EMA_INTS) {
			_max = ema[e*PRIXS + t-0*ema_ints[e].interv];
			_min = ema[e*PRIXS + t-0*ema_ints[e].interv];
			FOR(1, i, N_FLTR) {
				if (_max < ema[e*PRIXS + t-i*ema_ints[e].interv])
					_max = ema[e*PRIXS + t-i*ema_ints[e].interv];
				if (_min > ema[e*PRIXS + t-i*ema_ints[e].interv])
					_min = ema[e*PRIXS + t-i*ema_ints[e].interv];
			}
			FOR(0, i, N_FLTR) {
				normalisee[e*PRIXS*N_FLTR+t*N_FLTR+i] = (ema[e*PRIXS+t-i*ema_ints[e].interv]-_min)/(_max-_min);
			}
		};
	};

	FOR(DEPART, t, FIN) {
		FOR(0, e, EMA_INTS) {
			FOR(1, i, N_FLTR)
				dif_normalisee[e*PRIXS*N_FLTR+t*N_FLTR+i] = normalisee[e*PRIXS*N_FLTR+t*N_FLTR+i]-normalisee[e*PRIXS*N_FLTR+t*N_FLTR+i-1];
			dif_normalisee[e*PRIXS*N_FLTR+t*N_FLTR+N_FLTR+0] = 0.f;
		}
	}
};

float *          prixs__d = 0x0;
float *          macds__d = 0x0;
float *        volumes__d = 0x0;
float *          hight__d = 0x0;
float *            low__d = 0x0;
//
float *            ema__d = 0x0;
float *     normalisee__d = 0x0;
float * dif_normalisee__d = 0x0;

void charger_vram_nvidia() {
	CONTROLE_CUDA(hipMalloc((void**)&  prixs__d, sizeof(float) * PRIXS));
	CONTROLE_CUDA(hipMalloc((void**)&  macds__d, sizeof(float) * PRIXS));
	CONTROLE_CUDA(hipMalloc((void**)&volumes__d, sizeof(float) * PRIXS));
	CONTROLE_CUDA(hipMalloc((void**)&  hight__d, sizeof(float) * PRIXS));
	CONTROLE_CUDA(hipMalloc((void**)&    low__d, sizeof(float) * PRIXS));
	//
	CONTROLE_CUDA(hipMalloc((void**)&           ema__d, sizeof(float) * EMA_INTS * PRIXS *    1  ));
	CONTROLE_CUDA(hipMalloc((void**)&    normalisee__d, sizeof(float) * EMA_INTS * PRIXS * N_FLTR));
	CONTROLE_CUDA(hipMalloc((void**)&dif_normalisee__d, sizeof(float) * EMA_INTS * PRIXS * N_FLTR));
	//
	CONTROLE_CUDA(hipMemcpy(  prixs__d,   prixs, sizeof(float) * PRIXS, hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(  macds__d,   macds, sizeof(float) * PRIXS, hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(volumes__d, volumes, sizeof(float) * PRIXS, hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(  hight__d, volumes, sizeof(float) * PRIXS, hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(    low__d, volumes, sizeof(float) * PRIXS, hipMemcpyHostToDevice));
	//
	CONTROLE_CUDA(hipMemcpy(           ema__d,            ema, sizeof(float) * EMA_INTS * PRIXS *    1  , hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(    normalisee__d,     normalisee, sizeof(float) * EMA_INTS * PRIXS * N_FLTR, hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(dif_normalisee__d, dif_normalisee, sizeof(float) * EMA_INTS * PRIXS * N_FLTR, hipMemcpyHostToDevice));
};

void     liberer_cudamalloc() {
	CONTROLE_CUDA(hipFree(  prixs__d));
	CONTROLE_CUDA(hipFree(  macds__d));
	CONTROLE_CUDA(hipFree(volumes__d));
	CONTROLE_CUDA(hipFree(  hight__d));
	CONTROLE_CUDA(hipFree(    low__d));
	//
	CONTROLE_CUDA(hipFree(           ema__d));
	CONTROLE_CUDA(hipFree(    normalisee__d));
	CONTROLE_CUDA(hipFree(dif_normalisee__d));
};

void charger_tout() {
	printf("charger_les_prixs : ");      MESURER(charger_les_prixs());
	printf("calculer_ema_norm_diff : "); MESURER(calculer_ema_norm_diff());
	printf("charger_les_prixs : ");      MESURER(charger_vram_nvidia());
	printf("Méga-octés = %f Mo\n",
		(float)sizeof(float)*(PRIXS*3 + PRIXS*EMA_INTS*1 + PRIXS*EMA_INTS*N_FLTR + PRIXS*EMA_INTS*N_FLTR) / 1e6f
	);
};

void liberer_tout() {
	titre("Liberer tout");
	liberer_cudamalloc();
};